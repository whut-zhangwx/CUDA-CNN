#include "hip/hip_runtime.h"
#include "layer.h"

// Constructor
Layer::Layer(int M, int N, int O)
{
	this->M = M;
	this->N = N;
	this->O = O;

	float h_bias[N];
	float h_weight[N][M];

	output = NULL;
	preact = NULL;
	bias   = NULL;
	weight = NULL;

	for (int i = 0; i < N; ++i) {
		h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);
		/*h_bias[i] = 0.0f;*/

		for (int j = 0; j < M; ++j) {
			h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);
			/*h_weight[i][j] = 0.05f;*/
		}
	}

	hipMalloc(&output, sizeof(float) * O);
	hipMalloc(&preact, sizeof(float) * O);

	hipMalloc(&bias, sizeof(float) * N);

	hipMalloc(&weight, sizeof(float) * M * N);

	hipMalloc(&d_output, sizeof(float) * O);
	hipMalloc(&d_preact, sizeof(float) * O);
	hipMalloc(&d_weight, sizeof(float) * M * N);

	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);

	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
}

// Destructor
Layer::~Layer()
{
	hipFree(output);
	hipFree(preact);

	hipFree(bias);

	hipFree(weight);

	hipFree(d_output);
	hipFree(d_preact);
	hipFree(d_weight);
}

// Send data one row from dataset to the GPU
void Layer::setOutput(float *data)
{
	hipMemcpy(output, data, sizeof(float) * O, hipMemcpyHostToDevice);
}

// Reset GPU memory between iterations
void Layer::clear()
{
	hipMemset(output, 0x00, sizeof(float) * O);
	hipMemset(preact, 0x00, sizeof(float) * O);
}

void Layer::bp_clear()
{
	hipMemset(d_output, 0x00, sizeof(float) * O);
	hipMemset(d_preact, 0x00, sizeof(float) * O);
	hipMemset(d_weight, 0x00, sizeof(float) * M * N);
}

// sigmoid activation function
__device__ float step_function(float v)
{
	return 1 / (1 + exp(-v));
}

// apply sigmoid activation function
__global__ void apply_step_function(float *input, float *output, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		output[idx] = step_function(input[idx]);
	}
}

// difference of prediction and label
__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;
	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
	}
}

__global__ void apply_grad(float *output, float *grad, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		output[idx] += dt * grad[idx];
	}
}

__global__ void fp_preact_c1(float input[28][28], float preact[6][24][24], float weight[6][5][5])
{
	/*
	input: 28*28, channel: 1
	kernel: 6*5*5, channel: 6
	output: 6*24*24, channel: 6
	*/
	const int pos = blockIdx.x * blockDim.x + threadIdx.x; // tid: 0, 1, 2, ..., 64*64-1
	const int size = blockDim.x * gridDim.x; // 64*64: number of threads

	const int N = (5*5)*(6*24*24); // number of multiplications

	/*
	[i1][i2]: [0,0] ... [4,0]; [0,1] ... [4,1]; ......; [0,4] ... [4,4]
	[i3][i4][i5]: [0,0,0],...,[5,0,0] | [0,1,0],...,[5,1,0] | ... | [0,23,0],...,[5,23,0] | 
		            [0,0,1],...,[5,0,1] | [0,1,0],...,[5,1,1] | ... | [0,23,1],...,[5,23,1] | 
                ...
	              [0,0,23],...,[5,0,23] | [0,1,23],...,[5,1,23] | ... | [0,23,23],...,[5,23,23]
	*/
	/* N/size: the number of multiplications per thread */ 
	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 5);  // idx = n
		const int i2 = ((idx /= 5	) % 5);  // idx = n / 5
		const int i3 = ((idx /= 5	) % 6);  // idx = n / (5*5)
		const int i4 = ((idx /= 6	) % 24); // idx = n / (5*5*6)
		const int i5 = ((idx /= 24) % 24); // idx = n / (5*5*6*24)

		// https://docs.nvidia.com/cuda/cuda-c-programming-guide/#atomic-functions
		/*For example, atomicAdd() reads a word at some address in global or shared memory,
		adds a number to it, and writes the result back to the same address.
		Atomic functions can only be used in device functions.*/
		atomicAdd(&preact[i3][i4][i5], weight[i3][i1][i2] * input[i4 + i1][i5 + i2]);
	}
}

__global__ void fp_bias_c1(float preact[6][24][24], float bias[6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*24*24; // number of elements in preact
	/*
	[i1][i2][i3]: [0,0,0],...,[5,0,0] | [0,1,0],...,[5,1,0] | ... | [0,23,0],...,[5,23,0] | 
		            [0,0,1],...,[5,0,1] | [0,1,0],...,[5,1,1] | ... | [0,23,1],...,[5,23,1] | 
                ...
	              [0,0,23],...,[5,0,23] | [0,1,23],...,[5,1,23] | ... | [0,23,23],...,[5,23,23]
	*/

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);    // idx = n
		const int i2 = ((idx /= 6	) % 24);   // idx = n / 6
		const int i3 = ((idx /= 24	) % 24); // idx = n / (6*24)

		preact[i1][i2][i3] += bias[i1];
	}
}

// 6×24×24 --> 6×6×6 (kernal: 1×4×4, stride: 4)
__global__ void fp_preact_s1(float input[6][24][24], float preact[6][6][6], float weight[1][4][4])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 4*4*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 4);
		const int i2 = ((idx /= 4	) % 4);
		const int i3 = ((idx /= 4	) % 6);
		const int i4 = ((idx /= 6	) % 6);
		const int i5 = ((idx /= 6	) % 6);

		atomicAdd(&preact[i3][i4][i5], weight[0][i1][i2] * input[i3][i4 * 4 + i1][i5 * 4 + i2]);
	}
}

__global__ void fp_bias_s1(float preact[6][6][6], float bias[1])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 6);
		const int i3 = ((idx /= 6	) % 6);

		preact[i1][i2][i3] += bias[0];
	}
}

// fully connected layer
__global__ void fp_preact_f(float input[6][6][6], float preact[10], float weight[10][6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 10);
		const int i2 = ((idx /= 10	) % 6);
		const int i3 = ((idx /= 6	) % 6);
		const int i4 = ((idx /= 6	) % 6);

		atomicAdd(&preact[i1], weight[i1][i2][i3][i4] * input[i2][i3][i4]);
	}
}

__global__ void fp_bias_f(float preact[10], float bias[10])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		preact[idx] += bias[idx];
	}
}

__global__ void bp_weight_f(float d_weight[10][6][6][6], float d_preact[10], float p_output[6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 10);
		const int i2 = ((idx /= 10) % 6);
		const int i3 = ((idx /= 6	) % 6);
		const int i4 = ((idx /= 6	) % 6);
		// gradient: d_weight
		// upgrade: weight += d_weight
		d_weight[i1][i2][i3][i4] = d_preact[i1] * p_output[i2][i3][i4];
	}
}

__global__ void bp_bias_f(float bias[10], float d_preact[10])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		bias[idx] += dt * d_preact[idx];
	}
}

__global__ void bp_output_s1(float d_output[6][6][6], float n_weight[10][6][6][6], float nd_preact[10])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 10);
		const int i2 = ((idx /= 10	) % 6);
		const int i3 = ((idx /= 6	) % 6);
		const int i4 = ((idx /= 6	) % 6);

		atomicAdd(&d_output[i2][i3][i4], n_weight[i1][i2][i3][i4] * nd_preact[i1]);
	}
}

__global__ void bp_preact_s1(float d_preact[6][6][6], float d_output[6][6][6], float preact[6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 6);
		const int i3 = ((idx /= 6	) % 6);

		const float o = step_function(preact[i1][i2][i3]);

		d_preact[i1][i2][i3] = d_output[i1][i2][i3] * o * (1 - o);
	}
}

__global__ void bp_weight_s1(float d_weight[1][4][4], float d_preact[6][6][6], float p_output[6][24][24])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 1*4*4*6*6*6;
	const float d = pow(6.0f, 3.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 1);
		const int i2 = ((idx /= 1	) % 4);
		const int i3 = ((idx /= 4	) % 4);
		const int i4 = ((idx /= 4	) % 6);
		const int i5 = ((idx /= 6	) % 6);
		const int i6 = ((idx /= 6	) % 6);

		atomicAdd(&d_weight[i1][i2][i3], d_preact[i4][i5][i6] * p_output[i4][i5 * 4 + i2][i6 * 4 + i3]);
	}
}

__global__ void bp_bias_s1(float bias[1], float d_preact[6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*6*6;
	const float d = pow(6.0f, 3.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 6);
		const int i3 = ((idx /= 6	) % 6);

		atomicAdd(&bias[0], dt * d_preact[i1][i2][i3] / d);
	}
}

__global__ void bp_output_c1(float d_output[6][24][24], float n_weight[1][4][4], float nd_preact[6][6][6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 1*4*4*6*6*6;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 1);
		const int i2 = ((idx /= 1	) % 4);
		const int i3 = ((idx /= 4	) % 4);
		const int i4 = ((idx /= 4	) % 6);
		const int i5 = ((idx /= 6	) % 6);
		const int i6 = ((idx /= 6	) % 6);

		atomicAdd(&d_output[i4][i5 * 4 + i2][i6 * 4 + i3], n_weight[i1][i2][i3] * nd_preact[i4][i5][i6]);
	}
}

__global__ void bp_preact_c1(float d_preact[6][24][24], float d_output[6][24][24], float preact[6][24][24])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*24*24;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 24);
		const int i3 = ((idx /= 24	) % 24);

		const float o = step_function(preact[i1][i2][i3]);

		d_preact[i1][i2][i3] = d_output[i1][i2][i3] * o * (1 - o);
	}
}

__global__ void bp_weight_c1(float d_weight[6][5][5], float d_preact[6][24][24], float p_output[28][28])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*5*5*24*24;
	const float d = pow(24.0f, 2.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 5);
		const int i3 = ((idx /= 5	) % 5);
		const int i4 = ((idx /= 5	) % 24);
		const int i5 = ((idx /= 24	) % 24);

		atomicAdd(&d_weight[i1][i2][i3], d_preact[i1][i4][i5] * p_output[i4 + i2][i5 + i3] / d);
	}
}

__global__ void bp_bias_c1(float bias[6], float d_preact[6][24][24])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*24*24;
	const float d = pow(24.0f, 2.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 24);
		const int i3 = ((idx /= 24	) % 24);

		atomicAdd(&bias[i1], dt * d_preact[i1][i2][i3] / d);
	}
}
